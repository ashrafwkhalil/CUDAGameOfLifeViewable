#include "hip/hip_runtime.h"
﻿#include "G:\vs\repos\A3426\A3426\Dependencies\glew\include\GL\glew.h" // must be first
#include "G:\vs\repos\A3426\A3426\Dependencies\freeglut\include\GL\freeglut.h"
#include <iostream>
#include <string>
#include <thread>
using namespace std;
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.h"
const int size = 600;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t doWorkCUDA(int matrix[HEIGHT * WIDTH], int bufferMatrix[HEIGHT * WIDTH], unsigned int sizee);
void initMatrix(int matrix[HEIGHT * WIDTH], int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__device__ int xCompute(int i)
{
    return i % WIDTH;
}
__device__ int yCompute(int i)
{
    return i / WIDTH;
}
__device__ int xyCompute(int i, int j)
{
    return (j * WIDTH) + i;
}

__global__ void computeNextMatrixKernel(int matrix[HEIGHT * WIDTH], int bufferMatrix[HEIGHT * WIDTH])
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int surround[8];
    if (xCompute(i) > 0 && yCompute(i) > 0)
    {
        surround[0] = matrix[xyCompute(xCompute(i) - 1, yCompute(i) - 1)];
    }
    else
        surround[0] = 0;
    if (yCompute(i) > 0)
        surround[1] = matrix[xyCompute(xCompute(i), yCompute(i) - 1)];
    else
        surround[1] = 0;
    if (xCompute(i) < WIDTH && yCompute(i) > 0)
        surround[2] = matrix[xyCompute(xCompute(i) + 1, yCompute(i) - 1)];
    else
        surround[2] = 0;
    if (xCompute(i) < WIDTH)
        surround[3] = matrix[xyCompute(xCompute(i) + 1, yCompute(i))];
    else
        surround[3] = 0;
    if (xCompute(i) < WIDTH && yCompute(i) < HEIGHT)
        surround[4] = matrix[xyCompute(xCompute(i) + 1, yCompute(i) + 1)];
    else
        surround[4] = 0;
    if (yCompute(i) < HEIGHT)
        surround[5] = matrix[xyCompute(xCompute(i), yCompute(i) + 1)];
    else
        surround[5] = 0;
    if (xCompute(i) > 0 && yCompute(i) < HEIGHT)
        surround[6] = matrix[xyCompute(xCompute(i) - 1, yCompute(i) + 1)];
    else
        surround[6] = 0;
    if (xCompute(i) > 0)
        surround[7] = matrix[xyCompute(xCompute(i) - 1, yCompute(i))];
    else
        surround[7] = 0;

    int live[4];
    for (int ii = 0; ii < 4; ii++)
    {
        int counter = 0;
        for (int i = 0; i < 8; i++)
        {
            if (surround[i] == ii + 1)
            {
                counter++;
            }
        }
        live[ii] = counter;
    }
    if (matrix[i] != 0)
    {

        if (live[matrix[i] - 1] < 2)
        {

            bufferMatrix[i] = 0;
        }
        else if ((live[matrix[i] - 1] == 2) || (live[matrix[i] - 1] == 3))
        {

            bufferMatrix[i] = matrix[i];
        }
        else if (live[matrix[i] - 1] > 3)
        {

            bufferMatrix[i] = 0;
        }
    }
    else if (matrix[i] == 0)
    {
        int c = 0;
        int d = 0;
        for (int jj = 0; jj < 4; jj++)
        {
            if (live[jj] == 3)
            {
                c = jj;
                d = 1;
                break;
            }
        }
        if (d == 1)
            bufferMatrix[i] = c + 1;
        else
            bufferMatrix[i] = 0;
    }
}

hipError_t doWorkCUDA(int matrix[HEIGHT * WIDTH], int bufferMatrix[HEIGHT * WIDTH], unsigned int size)
{

    int *dev_matrix = 0;
    int *dev_bufferMatrix = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_matrix, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&dev_bufferMatrix, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_matrix, matrix, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy to device failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_bufferMatrix, bufferMatrix, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy to device failed!");
        goto Error;
    }

    computeNextMatrixKernel<<<768, 1024>>>(dev_matrix, dev_bufferMatrix);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching nextMatrix!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(bufferMatrix, dev_bufferMatrix, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy to host failed!");
        goto Error;
    }

Error:
    hipFree(dev_matrix);
    hipFree(dev_bufferMatrix);

    return cudaStatus;
}
